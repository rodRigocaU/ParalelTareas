#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "img2GC.h"
#include <math.h>
#include <stdio.h>
#include <iostream>
using namespace std;

/*
 -> To develop this homework -> Rodrigo Cayro
 i use two different techniques: one based in histograms and the second based in the formule of the book
*/

//tecchnique #1 based in histograms
__global__ void togrey(unsigned char* image, int* H);

//ALLOCATE MEMORY FOR THE PROCCESS
void calculate_grey_image(unsigned char* Image, int h, int w, int channels, int* histogram)
{
    unsigned char* DEV_img = NULL;
    int* DEV_h = NULL;
    hipMalloc((void**)&DEV_img, h * w * channels);
    hipMalloc((void**)&DEV_h, 256 * sizeof(int));

    //TRANSFER DATA FROM CPU TO GPU 
    hipMemcpy(DEV_img, Image, h * w * channels, hipMemcpyHostToDevice);
    hipMemcpy(DEV_img, Image, 256 * sizeof(int), hipMemcpyHostToDevice);

    //IMAGE HISTOGRAM
    dim3 Grid_image(h, w);
    togrey << <Grid_image, 1 >> >(DEV_img,DEV_h);

    //from GPU => CPU
    hipMemcpy(histogram, DEV_h, 256 * sizeof(int), hipMemcpyDeviceToHost);

    //free memory
    hipFree(DEV_h);
    hipFree(DEV_img);
}
__global__ void togrey(unsigned char* image, int* H)
{
    int x = blockIdx.x;
    int y = blockIdx.y;

    int Image_Idx = x + y * gridDim.x;
    atomicAdd(&H[image[Image_Idx]], 1);
}


//tecchnique #2 based in formule
__global__ void togrey2(unsigned char* image, unsigned char* output, int h, int w, int CHANNELS)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    //USING THE FORMULE WE COMPUTE FOR EACH PIXEL HIS GREY VERSION
    if (col < w && row < h)
    {
        int grey_offset = row * w + col;
        int rgb_offset = grey_offset * CHANNELS;

        unsigned char r = image[rgb_offset + 0];
        unsigned char g = image[rgb_offset + 1];
        unsigned char b = image[rgb_offset + 2];

        output[grey_offset] = r * 0.299f + g * 0.587f + b * 0.114f;
    }
}


void calculate_grey_image2(unsigned char* Image, unsigned char* output, int h, int w, int channels)
{
    unsigned char* DEV_img = NULL;
    unsigned char* DEV_output = NULL;

    hipMalloc((void**)&DEV_img, h * w * channels);
    hipMalloc((void**)&DEV_output, h * w);

    //TRANSFER DATA FROM CPU TO GPU 
    hipMemcpy(DEV_img, Image, h * w * channels, hipMemcpyHostToDevice);

    //WE DIVIDE THE GRID IN  16 PARTS
    //dim3 Grid_image((int)ceil(h / 16), (int)ceil(w / 16));
    dim3 Grid_image(w, h);
    //dim3 dimBlock(16, 16);
    togrey2 << <Grid_image, 1 >> > (DEV_img, DEV_output, h, w, channels);

    //TRANSFER DATA FROM GPU TO CPU 
    hipMemcpy(output, DEV_output, h * w, hipMemcpyDeviceToHost);


    //free memory
    hipFree(DEV_output);
    hipFree(DEV_img);

}

//tecchnique #2 based in formule
__global__ void toblur(unsigned char* image, unsigned char* output, int h, int w)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int BLUR_SIZE = 20;

    //USING THE FORMULE WE COMPUTE FOR EACH PIXEL HIS GREY VERSION
    if (col < w && row < h)
    {
        int pixVal = 0;
        int pixels = 0;

        for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow)
        {
            for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol)
            {
                int curRow = row + blurRow;
                int curCol = col + blurCol;
                if (curRow > -1 && curRow < h && curCol > -1 && curCol < w)
                {
                    pixVal += image[curRow * w + curCol];
                    pixels++;
                }
            }
        }
        output[row * w + col] = (unsigned char) (pixVal / pixels);

    }
}


void calculate_blur_image(unsigned char* Image, unsigned char* output, int h, int w)
{
    //cout << "ENTRE" << '\n';
    unsigned char* DEV_img = NULL;
    unsigned char* DEV_output = NULL;

    hipMalloc((void**)&DEV_img, h * w );
    hipMalloc((void**)&DEV_output, h * w );

    //TRANSFER DATA FROM CPU TO GPU 
    hipMemcpy(DEV_img, Image, h * w, hipMemcpyHostToDevice);

    //WE DIVIDE THE GRID IN  16 PARTS
    //dim3 Grid_image(w, h);
    dim3 Grid_image((int)ceil(w / 16), (int)ceil(h / 16));
    dim3 dimBlock(16, 16);
    toblur << <Grid_image, dimBlock >> > (DEV_img, DEV_output, h, w);

    //TRANSFER DATA FROM GPU TO CPU 
    hipMemcpy(output, DEV_output, h * w, hipMemcpyDeviceToHost);


    //free memory
    hipFree(DEV_output);
    hipFree(DEV_img);

}


